// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

/*! \file LJDriverPotentialPairGPU.cu
    \brief Defines the driver functions for computing all types of pair forces on the GPU
*/

#include "EvaluatorPairForceShiftedLJPlugin.h"
#include "AllDriverPotentialPairPluginGPU.cuh"

hipError_t gpu_compute_forceshiftedljplugintemp_forces(const pair_args_t& pair_args,
                                      const Scalar2 *d_params)
    {
    return gpu_compute_pair_forces<EvaluatorPairForceShiftedLJPlugin>(pair_args,
                                                    d_params);
    }


